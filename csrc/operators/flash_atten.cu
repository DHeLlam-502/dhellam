#include "hip/hip_runtime.h"
#include "ops.h"
#include "smctrl.h"
#define CHECK_DEVICE(x) TORCH_CHECK(x.is_cuda(), #x " must be on CUDA")
#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream, bool force_split_kernel=false) {
    FP16_SWITCH(!params.is_bf16, [&] {
        FWD_HEADDIM_SWITCH(params.d, [&] {
            if (params.num_splits <= 1 && !force_split_kernel) {  // If we don't set it num_splits == 0
                run_mha_fwd_<elem_type, kHeadDim>(params, stream);
            } else {
                run_mha_fwd_splitkv_dispatch<elem_type, kHeadDim>(params, stream);
            }
        });
    });
}

void run_mha_bwd(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    FP16_SWITCH(!params.is_bf16, [&] {
        if (params.d <= 32) {
            run_mha_bwd_<elem_type, 32>(params, stream, configure);
        } else if (params.d <= 64) {
            run_mha_bwd_<elem_type, 64>(params, stream, configure);
        } else if (params.d <= 96) {
            run_mha_bwd_<elem_type, 96>(params, stream, configure);
        } else if (params.d <= 128) {
            run_mha_bwd_<elem_type, 128>(params, stream, configure);
        } else if (params.d <= 160) {
            run_mha_bwd_<elem_type, 160>(params, stream, configure);
        } else if (params.d <= 192) {
            run_mha_bwd_<elem_type, 192>(params, stream, configure);
        } else if (params.d <= 224) {
          run_mha_bwd_<elem_type, 224>(params, stream, configure);
        } else if (params.d <= 256) {
          run_mha_bwd_<elem_type, 256>(params, stream, configure);
        }
    });
}

void set_params_fprop(Flash_fwd_params &params,
                      // sizes
                      const size_t b,
                      const size_t seqlen_q,
                      const size_t seqlen_k,
                      const size_t seqlen_q_rounded,
                      const size_t seqlen_k_rounded,
                      const size_t h,
                      const size_t h_k,
                      const size_t d,
                      const size_t d_rounded,
                      // device pointers
                      const at::Tensor q,
                      const at::Tensor k,
                      const at::Tensor v,
                      at::Tensor out,
                      void *cu_seqlens_q_d,
                      void *cu_seqlens_k_d,
                      void *seqused_k,
                      void *p_d,
                      void *softmax_lse_d,
                      float p_dropout,
                      float softmax_scale,
                      int window_size_left,
                      int window_size_right) {

    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.is_bf16 = q.dtype() == torch::kBFloat16;

    // Set the pointers and strides.
    params.q_ptr = q.data_ptr();
    params.k_ptr = k.data_ptr();
    params.v_ptr = v.data_ptr();
    // All stride are in elements, not bytes.
    params.q_row_stride = q.stride(-3);
    params.k_row_stride = k.stride(-3);
    params.v_row_stride = v.stride(-3);
    params.q_head_stride = q.stride(-2);
    params.k_head_stride = k.stride(-2);
    params.v_head_stride = v.stride(-2);
    params.o_ptr = out.data_ptr();
    params.o_row_stride = out.stride(-3);
    params.o_head_stride = out.stride(-2);

    if (cu_seqlens_q_d == nullptr) {
        params.q_batch_stride = q.stride(0);
        params.k_batch_stride = k.stride(0);
        params.v_batch_stride = v.stride(0);
        params.o_batch_stride = out.stride(0);
    }

    params.cu_seqlens_q = static_cast<int *>(cu_seqlens_q_d);
    params.cu_seqlens_k = static_cast<int *>(cu_seqlens_k_d);
    params.seqused_k = static_cast<int *>(seqused_k);

    // P = softmax(QK^T)
    params.p_ptr = p_d;

    // Softmax sum
    params.softmax_lse_ptr = softmax_lse_d;

    // Set the dimensions.
    params.b = b;
    params.h = h;
    params.h_k = h_k;
    params.h_h_k_ratio = h / h_k;
    params.seqlen_q = seqlen_q;
    params.seqlen_k = seqlen_k;
    params.seqlen_q_rounded = seqlen_q_rounded;
    params.seqlen_k_rounded = seqlen_k_rounded;
    params.d = d;
    params.d_rounded = d_rounded;

    // Set the different scale values.
    params.scale_softmax = softmax_scale;
    params.scale_softmax_log2 = softmax_scale * M_LOG2E;

    // Set this to probability of keeping an element to simplify things.
    params.p_dropout = 1.f - p_dropout;
    // Convert p from float to int so we don't have to convert the random uint to float to compare.
    // [Minor] We want to round down since when we do the comparison we use <= instead of <
    // params.p_dropout_in_uint = uint32_t(std::floor(params.p_dropout * 4294967295.0));
    // params.p_dropout_in_uint16_t = uint16_t(std::floor(params.p_dropout * 65535.0));
    params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
    params.rp_dropout = 1.f / params.p_dropout;
    params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
    TORCH_CHECK(p_dropout < 1.f);

    // Causal is the special case where window_size_right == 0 and window_size_left < 0.
    // Local is the more general case where window_size_right >= 0 or window_size_left >= 0.
    params.is_causal = window_size_left < 0 && window_size_right == 0;

    if (window_size_left < 0 && window_size_right >= 0) { window_size_left = seqlen_k; }
    if (window_size_left >= 0 && window_size_right < 0) { window_size_right = seqlen_k; }
    params.window_size_left = window_size_left;
    params.window_size_right = window_size_right;

    params.is_seqlens_k_cumulative = true;
}

void set_params_dgrad(Flash_bwd_params &params,
                      // sizes
                      const size_t b,
                      const size_t seqlen_q,
                      const size_t seqlen_k,
                      const size_t seqlen_q_rounded,
                      const size_t seqlen_k_rounded,
                      const size_t h,
                      const size_t h_k,
                      const size_t d,
                      const size_t d_rounded,
                      // device pointers
                      const at::Tensor q,
                      const at::Tensor k,
                      const at::Tensor v,
                      const at::Tensor out,
                      const at::Tensor dout,
                      at::Tensor dq,
                      at::Tensor dk,
                      at::Tensor dv,
                      void *cu_seqlens_q_d,
                      void *cu_seqlens_k_d,
                      void *dq_accum_d,
                      void *dk_accum_d,
                      void *dv_accum_d,
                      void *softmax_lse_d,
                      void *dsoftmax_sum_d,
                      float p_dropout,
                      float softmax_scale,
                      int window_size_left,
                      int window_size_right,
                      bool deterministic) {

    set_params_fprop(params,
                     b, seqlen_q, seqlen_k, seqlen_q_rounded, seqlen_k_rounded, h, h_k, d, d_rounded,
                     q, k, v, out,
                     cu_seqlens_q_d,
                     cu_seqlens_k_d,
                     nullptr,
                     nullptr,
                     softmax_lse_d,
                     p_dropout,
                     softmax_scale,
                     window_size_left,
                     window_size_right);

    // Set the pointers and strides.
    params.do_ptr = dout.data_ptr();
    params.do_row_stride = dout.stride(-3);
    params.do_head_stride = dout.stride(-2);
    params.dq_ptr = dq.data_ptr();
    params.dk_ptr = dk.data_ptr();
    params.dv_ptr = dv.data_ptr();
    params.dq_row_stride = dq.stride(-3);
    params.dk_row_stride = dk.stride(-3);
    params.dv_row_stride = dv.stride(-3);
    params.dq_head_stride = dq.stride(-2);
    params.dk_head_stride = dk.stride(-2);
    params.dv_head_stride = dv.stride(-2);

    if (cu_seqlens_q_d == nullptr) {
        params.do_batch_stride = dout.stride(0);
        params.dq_batch_stride = dq.stride(0);
        params.dk_batch_stride = dk.stride(0);
        params.dv_batch_stride = dv.stride(0);
    }

    params.dq_accum_ptr = dq_accum_d;
    params.dk_accum_ptr = dk_accum_d;
    params.dv_accum_ptr = dv_accum_d;

    // Softmax sum
    params.dsoftmax_sum = dsoftmax_sum_d;

    params.deterministic = deterministic;
}

std::vector<at::Tensor>
mha_varlen_fwd(const at::Tensor &q,  // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               const at::Tensor &k,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &v,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               c10::optional<at::Tensor> &out_, // total_q x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &cu_seqlens_q,  // b+1
               const at::Tensor &cu_seqlens_k,  // b+1
               c10::optional<at::Tensor> &seqused_k, // b. If given, only this many elements of each batch element's keys are used.
               c10::optional<at::Tensor> &alibi_slopes_, // num_heads or b x num_heads
               const int max_seqlen_q,
               const int max_seqlen_k,
               const float p_dropout,
               const float softmax_scale,
               const bool zero_tensors,
               const bool is_causal,
               int window_size_left,
               int window_size_right,
               const bool return_softmax,
               c10::optional<at::Generator> gen_ ,
               int math_sm_count, 
               const bool profiling,
               hipStream_t stream) {

    if (is_causal) { window_size_right = 0; }
    auto dprops = at::cuda::getCurrentDeviceProperties();
    // bool is_sm75 = dprops->major == 7 && dprops->minor == 5;
    bool is_sm8x = dprops->major == 8 && dprops->minor >= 0;
    bool is_sm90 = dprops->major == 9 && dprops->minor == 0;
    bool reverse = math_sm_count<0?true:false;
    math_sm_count = abs(math_sm_count);
    TORCH_CHECK(is_sm90 || is_sm8x, "FlashAttention only supports Ampere GPUs or newer.");
    // We will support Turing in the near future
    // TORCH_CHECK(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports Turing GPUs or newer.");

    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16,
                "FlashAttention only support fp16 and bf16 data type");
    if (q_dtype == torch::kBFloat16) {
        TORCH_CHECK(is_sm90 || is_sm8x, "bfloat16 is only supported on Ampere GPUs or newer");
    }
    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");
    TORCH_CHECK(cu_seqlens_q.dtype() == torch::kInt32, "cu_seqlens_q must have dtype int32");
    TORCH_CHECK(cu_seqlens_k.dtype() == torch::kInt32, "cu_seqlens_k must have dtype int32");

    CHECK_DEVICE(q); CHECK_DEVICE(k); CHECK_DEVICE(v);
    CHECK_DEVICE(cu_seqlens_q);
    CHECK_DEVICE(cu_seqlens_k);

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    CHECK_CONTIGUOUS(cu_seqlens_q);
    CHECK_CONTIGUOUS(cu_seqlens_k);

    const auto sizes = q.sizes();

    const int total_q = sizes[0];
    const int batch_size = cu_seqlens_q.numel() - 1;
    const int num_heads = sizes[1];
    const int head_size_og = sizes[2];
    const int total_k = k.size(0);
    const int num_heads_k = k.size(1);
    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size_og <= 256, "FlashAttention forward only supports head dimension at most 256");
    TORCH_CHECK(num_heads % num_heads_k == 0, "Number of heads in key/value must divide number of heads in query");

    if (window_size_left >= max_seqlen_k) { window_size_left = -1; }
    if (window_size_right >= max_seqlen_k) { window_size_right = -1; }

    CHECK_SHAPE(q, total_q, num_heads, head_size_og);
    CHECK_SHAPE(k, total_k, num_heads_k, head_size_og);
    CHECK_SHAPE(v, total_k, num_heads_k, head_size_og);
    CHECK_SHAPE(cu_seqlens_q, batch_size + 1);
    CHECK_SHAPE(cu_seqlens_k, batch_size + 1);
    if (seqused_k.has_value()){
        auto seqused_k_ = seqused_k.value();
        TORCH_CHECK(seqused_k_.dtype() == torch::kInt32, "seqused_k must have dtype int32");
        TORCH_CHECK(seqused_k_.is_cuda(), "seqused_k must be on CUDA device");
        TORCH_CHECK(seqused_k_.is_contiguous(), "seqused_k must be contiguous");
        CHECK_SHAPE(seqused_k_, batch_size);
    }

    at::Tensor q_padded, k_padded, v_padded;
    if (head_size_og % 8 != 0) {
        q_padded = torch::nn::functional::pad(q, torch::nn::functional::PadFuncOptions({0, 8 - head_size_og % 8}));
        k_padded = torch::nn::functional::pad(k, torch::nn::functional::PadFuncOptions({0, 8 - head_size_og % 8}));
        v_padded = torch::nn::functional::pad(v, torch::nn::functional::PadFuncOptions({0, 8 - head_size_og % 8}));
    } else {
        q_padded = q;
        k_padded = k;
        v_padded = v;
    }

    at::Tensor out;
    if (out_.has_value()) {
        out = out_.value();
        TORCH_CHECK(out.dtype() == q_dtype, "Output must have the same dtype as inputs");
        CHECK_DEVICE(out);
        TORCH_CHECK(out.stride(-1) == 1, "Output tensor must have contiguous last dimension");
        CHECK_SHAPE(out, total_q, num_heads, head_size_og);
        if (head_size_og % 8 != 0) { out = torch::empty_like(q_padded); }
    } else {
        out = torch::empty_like(q_padded);
    }

    auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
    const int head_size = round_multiple(head_size_og, 8);
    const int head_size_rounded = round_multiple(head_size, 32);
    const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
    const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)q.get_device()};

    auto opts = q.options();

    auto softmax_lse = torch::empty({batch_size, num_heads, max_seqlen_q}, opts.dtype(at::kFloat));
    at::Tensor p;
    // Only return softmax if there's dropout to reduce compilation time
    if (return_softmax) {
        TORCH_CHECK(p_dropout > 0.0f, "return_softmax is only supported when p_dropout > 0.0");
        p = torch::empty({ batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded }, opts);
    }

    if (zero_tensors) {
        out.zero_();
        softmax_lse.fill_(-std::numeric_limits<float>::infinity());
        if (return_softmax) {p.zero_();}
    }

    Flash_fwd_params params;
    set_params_fprop(params,
                     batch_size,
                     max_seqlen_q, max_seqlen_k,
                     seqlen_q_rounded, seqlen_k_rounded,
                     num_heads, num_heads_k,
                     head_size, head_size_rounded,
                     q_padded, k_padded, v_padded, out,
                     cu_seqlens_q.data_ptr(),
                     cu_seqlens_k.data_ptr(),
                     seqused_k.has_value() ? seqused_k.value().data_ptr() : nullptr,
                     return_softmax ? p.data_ptr() : nullptr,
                     softmax_lse.data_ptr(),
                     p_dropout,
                     softmax_scale,
                     window_size_left,
                     window_size_right);

    // number of times random will be generated per thread, to offset philox counter in thc random
    // state
    // We use a custom RNG that increases the offset by batch_size * nheads * 32.
    int64_t counter_offset = params.b * params.h * 32;
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
    // Forward kernel will populate memory with the seed and offset.
    params.rng_state = reinterpret_cast<uint64_t*>(rng_state.data_ptr());

    if (p_dropout > 0.0)  {
        auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
            gen_, at::cuda::detail::getDefaultCUDAGenerator());
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        params.philox_args = gen->philox_cuda_state(counter_offset);
    }

    if (alibi_slopes_.has_value()) {
        auto alibi_slopes = alibi_slopes_.value();
        TORCH_CHECK(alibi_slopes.dtype() == torch::kFloat32, "ALiBi slopes must have dtype fp32");
        CHECK_DEVICE(alibi_slopes);
        TORCH_CHECK(alibi_slopes.stride(-1) == 1, "ALiBi slopes tensor must have contiguous last dimension");
        TORCH_CHECK(alibi_slopes.sizes() == torch::IntArrayRef({num_heads}) || alibi_slopes.sizes() == torch::IntArrayRef({batch_size, num_heads}));
        params.alibi_slopes_ptr = alibi_slopes.data_ptr();
        params.alibi_slopes_batch_stride = alibi_slopes.dim() == 2 ? alibi_slopes.stride(0) : 0;
    } else {
        params.alibi_slopes_ptr = nullptr;
    }
    float elapsed_time = 0; // ms
    hipEvent_t start, stop;
    if (max_seqlen_k > 0) {
        // auto stream = at::cuda::getCurrentCUDAStream().stream();
        if(profiling){
            DHELLAM_CHECK_CUDA(hipEventCreate(&start));
            DHELLAM_CHECK_CUDA(hipEventCreate(&stop));
            DHELLAM_CHECK_CUDA(hipEventRecord(start, stream));
        }
        SET_SM_COUNT(static_cast<unsigned int>(math_sm_count), reverse);
        run_mha_fwd(params, stream);
        if(profiling){
            DHELLAM_CHECK_CUDA(hipEventRecord(stop,stream));
            DHELLAM_CHECK_CUDA(hipEventSynchronize(stop));
            DHELLAM_CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
            DHELLAM_CHECK_CUDA(hipEventDestroy(start));
            DHELLAM_CHECK_CUDA(hipEventDestroy(stop));

        }
    } else {
        // If seqlen_k == 0, then we have an empty tensor. We need to set the output to 0.
        out.zero_();
        softmax_lse.fill_(std::numeric_limits<float>::infinity());
    }

    at::Tensor out_padded = out;
    if (head_size_og % 8 != 0) {
        out = out.index({"...", torch::indexing::Slice(torch::indexing::None, head_size_og)});
        if (out_.has_value()) { out_.value().copy_(out); }
    }

    if(profiling){
        auto pass_time_tensor = torch::tensor({elapsed_time}, torch::dtype(torch::kFloat32));
        return {out, q_padded, k_padded, v_padded, out_padded, softmax_lse, p, rng_state, pass_time_tensor};
    }
    return {out, q_padded, k_padded, v_padded, out_padded, softmax_lse, p, rng_state};
}

std::vector<at::Tensor>
pymha_varlen_fwd(const at::Tensor &q,  // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               const at::Tensor &k,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &v,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &cu_seqlens_q,  // b+1
               const at::Tensor &cu_seqlens_k,  // b+1
               const int max_seqlen_q,
               const int max_seqlen_k,
               const float p_dropout,
               const float softmax_scale,
               const bool is_causal,
               const bool return_softmax,
               const int math_sm_count,
               const bool profiling){
    c10::optional<at::Tensor> out_ = std::nullopt;
    c10::optional<at::Tensor> seqused_k= std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_= std::nullopt;
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    return mha_varlen_fwd(
        q,
        k,
        v,
        out_,
        cu_seqlens_q,
        cu_seqlens_k,
        seqused_k,
        alibi_slopes_,
        max_seqlen_q,
        max_seqlen_k,
        p_dropout,
        softmax_scale,
        false,
        is_causal,
        -1,
        0,
        return_softmax,
        std::nullopt,
        math_sm_count,
        profiling,
        stream
    );
}

std::vector<at::Tensor>
mha_varlen_bwd(const at::Tensor &dout,  // total_q x num_heads, x head_size
               const at::Tensor &q,   // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               const at::Tensor &k,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &v,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &out,   // total_q x num_heads x head_size
               const at::Tensor &softmax_lse,     // b x h x s   softmax logsumexp
               c10::optional<at::Tensor> &dq_,   // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               c10::optional<at::Tensor> &dk_,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               c10::optional<at::Tensor> &dv_,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &cu_seqlens_q,  // b+1
               const at::Tensor &cu_seqlens_k,  // b+1
               c10::optional<at::Tensor> &alibi_slopes_, // num_heads or b x num_heads
               const int max_seqlen_q,
               const int max_seqlen_k,          // max sequence length to choose the kernel
               const float p_dropout,         // probability to drop
               const float softmax_scale,
               const bool zero_tensors,
               const bool is_causal,
               int window_size_left,
               int window_size_right,
               const bool deterministic,
               c10::optional<at::Generator> gen_,
               c10::optional<at::Tensor> &rng_state,
               int math_sm_count,
               const bool profiling,
               hipStream_t stream) {

    if (is_causal) { window_size_right = 0; }
    auto dprops = at::cuda::getCurrentDeviceProperties();
    // bool is_sm75 = dprops->major == 7 && dprops->minor == 5;
    bool is_sm8x = dprops->major == 8 && dprops->minor >= 0;
    bool is_sm80 = dprops->major == 8 && dprops->minor == 0;
    bool is_sm90 = dprops->major == 9 && dprops->minor == 0;
    bool reverse = math_sm_count<0?true:false;
    math_sm_count = abs(math_sm_count);
    TORCH_CHECK(is_sm90 || is_sm8x, "FlashAttention only supports Ampere GPUs or newer.");
    // We will support Turing in the near future
    // TORCH_CHECK(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports Turing GPUs or newer.");
    bool is_dropout = p_dropout > 0.0;
    //auto stream = at::cuda::getCurrentCUDAStream().stream();

    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16,
                "FlashAttention only support fp16 and bf16 data type");
    if (q_dtype == torch::kBFloat16) {
        TORCH_CHECK(is_sm90 || is_sm8x, "bfloat16 is only supported on Ampere GPUs or newer");
    }
    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");
    TORCH_CHECK(out.dtype() == q_dtype, "query and out must have the same dtype");
    TORCH_CHECK(dout.dtype() == q_dtype, "query and dout must have the same dtype");
    TORCH_CHECK(cu_seqlens_q.dtype() == torch::kInt32, "cu_seqlens_q must have dtype int32");
    TORCH_CHECK(cu_seqlens_k.dtype() == torch::kInt32, "cu_seqlens_k must have dtype int32");

    CHECK_DEVICE(q); CHECK_DEVICE(k); CHECK_DEVICE(v);
    CHECK_DEVICE(out); CHECK_DEVICE(dout); CHECK_DEVICE(softmax_lse);
    CHECK_DEVICE(cu_seqlens_q); CHECK_DEVICE(cu_seqlens_k);

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(out.stride(-1) == 1, "out tensor must have contiguous last dimension");
    TORCH_CHECK(dout.stride(-1) == 1, "dout tensor must have contiguous last dimension");
    CHECK_CONTIGUOUS(cu_seqlens_q);
    CHECK_CONTIGUOUS(cu_seqlens_k);

    const auto sizes = q.sizes();

    const int total_q = sizes[0];
    const int batch_size = cu_seqlens_q.numel() - 1;
    const int num_heads = sizes[1];
    const int head_size_og = dout.size(2);
    const int head_size = sizes[2];
    const int total_k = k.size(0);
    const int num_heads_k = k.size(1);
    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size % 8 == 0, "head_size should be a multiple of 8");
    TORCH_CHECK(head_size <= 256, "FlashAttention backward only supports head dimension at most 256");
    if (head_size > 192) {
        TORCH_CHECK(is_sm80 || is_sm90, "FlashAttention backward for head dim > 192 requires A100/A800 or H100/H800");
    }
    TORCH_CHECK(num_heads % num_heads_k == 0, "Number of heads in key/value must divide number of heads in query");

    auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
    const int head_size_rounded = round_multiple(head_size, 32);
    const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
    const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

    TORCH_CHECK(head_size == round_multiple(head_size_og, 8), "head_size must be head_size_og rounded to a multiple of 8");

    if (window_size_left >= max_seqlen_k) { window_size_left = -1; }
    if (window_size_right >= max_seqlen_k) { window_size_right = -1; }

    CHECK_SHAPE(q, total_q, num_heads, head_size);
    CHECK_SHAPE(k, total_k, num_heads_k, head_size);
    CHECK_SHAPE(v, total_k, num_heads_k, head_size);
    CHECK_SHAPE(out, total_q, num_heads, head_size);
    CHECK_SHAPE(dout, total_q, num_heads, head_size_og);
    CHECK_SHAPE(cu_seqlens_q, batch_size + 1);
    CHECK_SHAPE(cu_seqlens_k, batch_size + 1);

    at::Tensor dq, dk, dv;
    if (dq_.has_value()) {
        dq = dq_.value();
        TORCH_CHECK(dq.dtype() == q_dtype, "dq must have the same dtype as q");
        CHECK_DEVICE(dq);
        TORCH_CHECK(dq.stride(-1) == 1, "dq must have contiguous last dimension");
        CHECK_SHAPE(dq, total_q, num_heads, head_size);
    } else {
        dq = torch::empty_like(q);
    }
    if (dk_.has_value()) {
        dk = dk_.value();
        TORCH_CHECK(dk.dtype() == q_dtype, "dk must have the same dtype as q");
        CHECK_DEVICE(dk);
        TORCH_CHECK(dk.stride(-1) == 1, "dk must have contiguous last dimension");
        CHECK_SHAPE(dk, total_k, num_heads_k, head_size);
    } else {
        dk = torch::empty_like(k);
    }
    if (dv_.has_value()) {
        dv = dv_.value();
        TORCH_CHECK(dv.dtype() == q_dtype, "dv must have the same dtype as q");
        CHECK_DEVICE(dv);
        TORCH_CHECK(dv.stride(-1) == 1, "dv must have contiguous last dimension");
        CHECK_SHAPE(dv, total_k, num_heads_k, head_size);
    } else {
        dv = torch::empty_like(k);
    }

    at::Tensor dout_padded;
    if (head_size_og % 8 != 0) {
        dout_padded = torch::nn::functional::pad(dout, torch::nn::functional::PadFuncOptions({0, 8 - head_size_og % 8}));
    } else {
        dout_padded = dout;
    }

    // bool loop = max_seqlen_k > blocksize_c;
    // TODO: change later, for now set to true for simplicity
    bool loop = true;

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)q.get_device()};

    auto opts = q.options();
    auto softmax_d = torch::empty({batch_size, num_heads, seqlen_q_rounded}, opts.dtype(at::kFloat));
    at::Tensor dq_accum;
    if (loop) {
        // We don't want to allocate dq_accum of size (batch, seqlen_q_rounded, num_heads, head_size_rounded)
        // because that would be too large if there is a very long sequence and the rest of the sequences are short.
        // Instead, we allocate dq_accum of size (total_q + 128 * batch, num_heads, head_size_rounded).
        // Note that 128 is the max block size on the seqlen_q dimension.
        // For dQ, the i-th sequence is stored in indices from cu_seqlens[i] + 128 * i to
        // cu_seqlens[i + 1] * 128 * i - 1. This ensures that the i-th sequence and (i + 1)-th sequence will
        // be at least 128 apart. It's ok for us to do atomicAdds up to 128 rows beyond what we're normally
        // allowed to do. So we won't have to do any bound checking, and performance should stay the same.
        if (!deterministic) {
            dq_accum = torch::empty({total_q + 128 * batch_size, num_heads, head_size_rounded}, opts.dtype(at::kFloat));
        } else {
            const int nsplits = (dprops->multiProcessorCount + batch_size * num_heads - 1) / (batch_size * num_heads);
            dq_accum = torch::zeros({nsplits, total_q + 128 * batch_size, num_heads, head_size_rounded}, opts.dtype(at::kFloat));
        }
    }

    at::Tensor dk_expanded, dv_expanded;
    if (num_heads_k != num_heads) {  // MQA / GQA
        dk_expanded = torch::empty({total_k, num_heads, head_size}, opts);
        dv_expanded = torch::empty({total_k, num_heads, head_size}, opts);
    } else {
        dk_expanded = dk;
        dv_expanded = dv;
    }

    if( zero_tensors ) {
        dq.zero_();
        dk_expanded.zero_();
        dv_expanded.zero_();
        softmax_d.zero_();
    }

    Flash_bwd_params params;

    set_params_dgrad(params,
                     batch_size,
                     max_seqlen_q, max_seqlen_k,
                     seqlen_q_rounded, seqlen_k_rounded,
                     num_heads, num_heads_k,
                     head_size, head_size_rounded,
                     q, k, v, out,
                     dout_padded, dq, dk_expanded, dv_expanded,
                     cu_seqlens_q.data_ptr(),
                     cu_seqlens_k.data_ptr(),
                     loop ? dq_accum.data_ptr() : nullptr,
                     nullptr,
                     nullptr,
                     softmax_lse.data_ptr(),
                     softmax_d.data_ptr(),
                     p_dropout,
                     softmax_scale,
                     window_size_left,
                     window_size_right,
                     deterministic);
    params.dq_accum_split_stride = !deterministic ? 0 : dq_accum.stride(0);

    auto launch = &run_mha_bwd;
    // launch(params, stream, /*configure=*/true);

    auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
        gen_, at::cuda::detail::getDefaultCUDAGenerator());

    // We use a custom RNG that increases the offset by batch_size * nheads * 32.
    int64_t counter_offset = params.b * params.h * 32;

    if ( rng_state.has_value() ) {
        params.rng_state = reinterpret_cast<uint64_t*>(rng_state.value().data_ptr());
    } else if( is_dropout ) {
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        params.philox_args = gen->philox_cuda_state(counter_offset);
        auto seeds = at::cuda::philox::unpack(params.philox_args);
        params.rng_state[0] = std::get<0>(seeds);
        params.rng_state[1] = std::get<1>(seeds);
    }

    if (alibi_slopes_.has_value()) {
        auto alibi_slopes = alibi_slopes_.value();
        TORCH_CHECK(alibi_slopes.dtype() == torch::kFloat32, "ALiBi slopes must have dtype fp32");
        CHECK_DEVICE(alibi_slopes);
        TORCH_CHECK(alibi_slopes.stride(-1) == 1, "ALiBi slopes tensor must have contiguous last dimension");
        TORCH_CHECK(alibi_slopes.sizes() == torch::IntArrayRef({num_heads}) || alibi_slopes.sizes() == torch::IntArrayRef({batch_size, num_heads}));
        params.alibi_slopes_ptr = alibi_slopes.data_ptr();
        params.alibi_slopes_batch_stride = alibi_slopes.dim() == 2 ? alibi_slopes.stride(0) : 0;
    } else {
        params.alibi_slopes_ptr = nullptr;
    }
    float elapsed_time = 0; // ms
    hipEvent_t start, stop;
    if (max_seqlen_q > 0) {
        if(profiling){
            DHELLAM_CHECK_CUDA(hipEventCreate(&start));
            DHELLAM_CHECK_CUDA(hipEventCreate(&stop));
            DHELLAM_CHECK_CUDA(hipEventRecord(start, stream));
        }
        SET_SM_COUNT(static_cast<unsigned int>(math_sm_count), reverse);
        launch(params, stream, /*configure=*/false);
        if(profiling){
            DHELLAM_CHECK_CUDA(hipEventRecord(stop,stream));
            DHELLAM_CHECK_CUDA(hipEventSynchronize(stop));
            DHELLAM_CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
            DHELLAM_CHECK_CUDA(hipEventDestroy(start));
            DHELLAM_CHECK_CUDA(hipEventDestroy(stop));

        }
    } else {
        // If seqlen_q == 0, then we have an empty tensor. We need to set the output to 0.
        dk_expanded.zero_();
        dv_expanded.zero_();
        softmax_d.zero_();
    }
    
    // For MQA/GQA we need to sum dK and dV across the groups
    if (num_heads_k != num_heads) {
        at::sum_out(dk, at::reshape(dk_expanded, {total_k, num_heads_k, num_heads / num_heads_k, head_size}), {2});
        at::sum_out(dv, at::reshape(dv_expanded, {total_k, num_heads_k, num_heads / num_heads_k, head_size}), {2});
    }
    if (head_size_og % 8 != 0) {
        dq = dq.index({"...", torch::indexing::Slice(torch::indexing::None, head_size_og)});
        dk = dk.index({"...", torch::indexing::Slice(torch::indexing::None, head_size_og)});
        dv = dv.index({"...", torch::indexing::Slice(torch::indexing::None, head_size_og)});
    }

    if(profiling){
        auto pass_time_tensor = torch::tensor({elapsed_time}, torch::dtype(torch::kFloat32));
        return { dq, dk, dv, softmax_d, pass_time_tensor};
    }
    return { dq, dk, dv, softmax_d };
}


std::vector<at::Tensor>
pymha_varlen_bwd(const at::Tensor &dout,  // total_q x num_heads, x head_size
               const at::Tensor &q,   // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               const at::Tensor &k,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &v,   // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &out,   // total_q x num_heads x head_size
               const at::Tensor &softmax_lse,     // b x h x s   softmax logsumexp
               const at::Tensor &cu_seqlens_q,  // b+1
               const at::Tensor &cu_seqlens_k,  // b+1
               const int max_seqlen_q,
               const int max_seqlen_k,          // max sequence length to choose the kernel
               const float p_dropout,         // probability to drop
               const float softmax_scale,
               const bool is_causal,
               c10::optional<at::Tensor> &rng_state,
               const bool deterministic,
               const int math_sm_count,
               const bool profiling,
               uint64_t stream_ptr){
    c10::optional<at::Tensor> dq_ = std::nullopt;
    c10::optional<at::Tensor> dk_ = std::nullopt;
    c10::optional<at::Tensor> dv_ = std::nullopt;
    c10::optional<at::Tensor> alibi_slopes_ = std::nullopt;
    //c10::optional<at::Tensor> rng_state_ = std::nullopt;
    auto stream = reinterpret_cast<hipStream_t>(stream_ptr);
    return mha_varlen_bwd(
        dout,
        q,
        k,
        v,
        out,
        softmax_lse,
        dq_,
        dk_,
        dv_,
        cu_seqlens_q,
        cu_seqlens_k,
        alibi_slopes_,
        max_seqlen_q,
        max_seqlen_k,
        p_dropout,
        softmax_scale,
        false,
        is_causal,
        -1,
        0,
        deterministic,
        std::nullopt,
        rng_state,
        math_sm_count,
        profiling,
        stream
    );
}

